#include <iostream>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <chrono>

using namespace std;


/// With k20m and k40m GPUs banks are organized in sets of 8 bytes,
/// for this reason, conflicts happen when accesses to doubles fall on the
/// same bank
__global__ void MyKernelHomogeneos(unsigned long long * time) {
    const unsigned sharedSize = 4096;
    __shared__ double shared[sharedSize];
    unsigned long long startTime;
    unsigned long long finishTime;

    // const int idx = 0; //perform a broadcast
    // const int idx = blockIdx.x; // perform a broadcast
    // const int idx = threadIdx.x; // no bank conflict - each therad access different bank
    // const int idx = threadIdx.x*2; // bank conflict starts
    // const int idx = threadIdx.x*32; // worst bank conflict - all threads access same bank
    // const int idx = threadIdx.x*128; // same worst bank conflict

    const int idx = threadIdx.x*2; // current test
    if (idx < sharedSize) {

    // time the access an homogeneous array
    startTime = clock();
    shared[idx]++;
    finishTime = clock();

    time[threadIdx.x] = (finishTime - startTime);
    }
}

int main(int argc, char const *argv[])
{
    const unsigned nThreads = 32;

    unsigned long long time[nThreads];
    unsigned long long * d_time;

    hipMalloc(&d_time, sizeof(unsigned long long)*nThreads);

    const unsigned long long overhead = 0;
    for (int r = 0; r < 10; r++)
    {
        hipDeviceSetCacheConfig(hipFuncCachePreferShared);
        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        MyKernelHomogeneos<<< 1,nThreads >>>(d_time);
        hipMemcpy(&time, d_time, sizeof(unsigned long long)*nThreads, hipMemcpyDeviceToHost);

        cout << "Time:\t";
        for (int i = 0; i < nThreads; i++)
        {
         cout<<(time[i]-overhead)/32<<"\t";
        }
        cout << endl<<endl;
    }
    
    hipFree(d_time);
    hipDeviceReset();
    return 0;
}

