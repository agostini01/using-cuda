#include "hip/hip_runtime.h"
#include <iostream>
#include "FindClosestGPU.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

int FindClosestGPU (float3* points, int* indices, int count) {

    // Allocate and copy to device
    float3 * dev_points;
    int * dev_indices;

    if (hipMalloc(&dev_points, sizeof(float3)*count) != hipSuccess)
    {
        cout<< "Could not allocate dev_points" << endl;
        return 1;
    }
    if (hipMalloc(&dev_indices, sizeof(int)*count) != hipSuccess)
    {
        cout<< "Could not allocate dev_indices" << endl;
        hipFree(dev_points);
        return 1;
    }
    if (hipMemcpy(dev_points, points, sizeof(float3) * count, hipMemcpyHostToDevice)!=hipSuccess)
    {
        cout<< "Could not copy d_a" << endl;
        hipFree(dev_points);
        hipFree(dev_indices);
        return 1;
    }

    // Invoke kernel
    FindClosest<<<(count/32)+1, 32>>>(dev_points, dev_indices, count);

    // Wait for kernel

    // Copy data back

    return 0;
}

__global__ void FindClosest (float3* points, int* indices, int count) {
    if (count <= 1) return;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < count)
    {

        float3 thisPoint = points[idx];
        float distToClosest = 3.40282e38f; // float.max value for initial dist

        for (int i = 0; i < count; i++)
        {
            if (i == idx) continue;
            float dist = 
                (thisPoint.x - points[i].x) *
                (thisPoint.x - points[i].x) +
                (thisPoint.y - points[i].y) *
                (thisPoint.y - points[i].y) +
                (thisPoint.z - points[i].z) *
                (thisPoint.z - points[i].z)
            ;
            if(dist < distToClosest) {
                distToClosest = dist;
                indices[idx] = i;
            }
        }
    }
}