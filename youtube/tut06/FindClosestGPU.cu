#include "hip/hip_runtime.h"
#include <iostream>
#include "FindClosestGPU.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

int FindClosestGPU (float3* points, int* indices, int count) {

    // Allocate and copy to device
    float3 * dev_points;
    int * dev_indices;

    if (hipMalloc(&dev_points, sizeof(float3)*count) != hipSuccess)
    {
        cout<< "Could not allocate dev_points" << endl;
        return 1;
    }
    if (hipMalloc(&dev_indices, sizeof(int)*count) != hipSuccess)
    {
        cout<< "Could not allocate dev_indices" << endl;
        hipFree(dev_points);
        return 1;
    }
    if (hipMemcpy(dev_points, points, sizeof(float3) * count, hipMemcpyHostToDevice)!=hipSuccess)
    {
        cout<< "Could not copy d_a" << endl;
        hipFree(dev_points);
        hipFree(dev_indices);
        return 1;
    }

    // Invoke kernel
    FindClosestOpt<<<(count/512)+1, 512>>>(dev_points, dev_indices, count);

    // Copy data back
    if (hipMemcpy(indices, dev_indices, sizeof(int) * count, hipMemcpyDeviceToHost)!=hipSuccess)
    {
        cout<< "Could not copy back from device" << endl;
        hipFree(dev_points);
        hipFree(dev_indices);
        delete[] dev_points;
        delete[] dev_indices;
        return 1;
    }

    // Cleanup
    hipFree(dev_points);
    hipFree(dev_indices);

    return 0;
}

__global__ void FindClosest (float3* points, int* indices, int count) {
    if (count <= 1) return;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < count)
    {
        float3 thisPoint = points[idx];
        float distToClosest = 3.40282e38f; // float.max value for initial dist

        for (int i = 0; i < count; i++)
        {
            if (i == idx) continue;
            float dist = 
                (thisPoint.x - points[i].x) *
                (thisPoint.x - points[i].x) +
                (thisPoint.y - points[i].y) *
                (thisPoint.y - points[i].y) +
                (thisPoint.z - points[i].z) *
                (thisPoint.z - points[i].z)
            ;
            if(dist < distToClosest) {
                distToClosest = dist;
                indices[idx] = i;
            }
        }
    }
}

__global__ void FindClosestOpt (float3* points, int* indices, int count) {
    if (count <= 1) return;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < count)
    {

        float3 thisPoint = points[idx];
        float distToClosest = 3.40282e38f; // float.max value for initial dist
        int tmp;

        for (int i = 0; i < count; i++)
        {
            if (i == idx) continue;
            float dist = 
                (thisPoint.x - points[i].x) *
                (thisPoint.x - points[i].x) +
                (thisPoint.y - points[i].y) *
                (thisPoint.y - points[i].y) +
                (thisPoint.z - points[i].z) *
                (thisPoint.z - points[i].z)
            ;
            if(dist < distToClosest) {
                distToClosest = dist;
                tmp = i;
            }
        }
        indices[idx] = tmp;
    }
}